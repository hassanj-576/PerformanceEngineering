#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <string>
using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::string;
using std::setprecision;

__global__ void distance(const int nodeNumber,const struct Nodes * inputData) {
		//CODE MOTION

		unsigned int first = (blockIdx.x * blockDim.x) + threadIdx.x;
		unsigned int second = (blockIdx.y * blockDim.y) + threadIdx.y;
		if(first<nodeNumber && second<nodeNumber){
			int x = nodes[first].x;
			int y = nodes[first].y;
			double sq1 = (x-nodes[second].x)*(x-nodes[second].x);
			double sq2 = (y-nodes[second].y)*(y-nodes[second].y);
			nodes[first].neighbourDistance[second]=sq1 + sq2;
			nodes[first].neighbourID[second]=second;
		}

	
}

void distanceMain(const int nodeNumber, const struct Nodes * inputNodes) {
	   //Memory Timer		    
	NSTimer memoryTimer("MemoryTimer", false, false);
		//Creating variables to be used by Kernel
	struct Nodes * d_in;
	//unsigned char* d_out;
	//Allocating memory on Device
	hipMalloc((void**) &d_in, (nodeNumber*sizeof(struct Nodes)));
	//hipMalloc((void**) &d_out,width*height);
	memoryTimer.start();
	//Copying from Host to device 
	hipMemcpy(d_in, inputNodes, (nodeNumber*sizeof(struct Nodes)), hipMemcpyHostToDevice);
	memoryTimer.stop();
	 //Number of blocks is (width/16)+1 and (height/16)+1. We add 1 to each in case the width or height is not perfectly divisible by 1. We cater for any extra 
	 // threads we get in the kernel
	dim3 gridSize((width/16)+1,(height/16)+1);
		//16x16 Threads per block (utalizing 256 threads per block)
	dim3 blockSize(16,16);
	//Kernel Timer
	NSTimer kernelTimer = NSTimer("darker", false, false);
	kernelTimer.start();
		// Starting Kernel
	greyScale<<< gridSize, blockSize >>>(width,height,d_in, d_out);
	hipDeviceSynchronize();
	kernelTimer.stop();
			  
	memoryTimer.start();
	// Copying Resylts back from Device To Host
	hipMemcpy(inputNodes, d_in, (nodeNumber*sizeof(struct Nodes)), hipMemcpyDeviceToHost);
	memoryTimer.stop();
	//Printing Time
	cout << "Kernel (s): \t"<<fixed <<setprecision(6)<< kernelTimer.getElapsed() << endl;
  	cout << "Memory (s): \t"<<fixed <<setprecision(6)<< memoryTimer.getElapsed() << endl;


}