#include "hip/hip_runtime.h"
__global__ void histogram1DCuda(const int width, const int height, const unsigned char * inputImage, unsigned int * histogram){

	for (first=0;first<nodeNumber;first++){
		//CODE MOTION
		int x = nodes[first].x;
		int y = nodes[first].y;
		double max=DBL_MAX;
		int maxIndex=0;
		int empty=0;
		double *n_distance = nodeDistance[first].neighbourDistance;
		int *id = nodeDistance[first].neighbourID;
		//printf("new element : %d\n", first );

		for(second=0;second<nodeNumber;second++){
			// ORIGINAL
			// double sq1 = (nodes[first].x-nodes[second].x)*(nodes[first].x-nodes[second].x);
			// double sq2 = (nodes[first].y-nodes[second].y)*(nodes[first].y-nodes[second].y);
			//CODE MOTION
			double x1= x-nodes[second].x;
			double y1 = y-nodes[second].y;
			//double x1= 0.0;
			//double y1 = 0.0;
			double sq1=x1*x1;
			double sq2 = y1*y1;
			double distance =sq1+sq2;
			//printf(" distance :%f,  empty : %d\n",distance, empty);
			if((distance<max) || (empty < N)){
				
				if(empty==N){
					//printf("%s:%d\n",__FUNCTION__,__LINE__ );
					n_distance[maxIndex]=distance;
					id[maxIndex] = second;

				}else{
					//printf("%s:%d\n",__FUNCTION__,__LINE__ );
					n_distance[empty]=distance;
					id[empty] = second;
				//	nodes[first].neighbourDistance[empty]=distance;
				//	nodes[first].neighbourID[empty] = second;
					empty++;
				}
				
				max = 0.00;

				for(int i=0;i<empty;i++){
					if (n_distance[i] > max) {
						//max = nodes[first].neighbourDistance[i];
						max = n_distance[i];
						maxIndex = i;
					}
				}

				//printf("maxIndex : %d and max element %f at line %d\n", maxIndex, max,  __LINE__);
			}
			// double sq1 = (x-nodes[second].x)*(x-nodes[second].x);
			// double sq2 = (y-nodes[second].y)*(y-nodes[second].y);
				
			// nodeDistance[first].neighbourDistance[second]=sq1 + sq2;
			// nodeDistance[first].neighbourID[second]=second;
			

			// distance[first][second]=sq1+sq2;
			//distance[first][second]=sq1 + sq2;
			// nodes[first].neighbourID[second]=second;
			// nodes[first].neighbourDistance[second]=sq1 + sq2;
		}

	}
}