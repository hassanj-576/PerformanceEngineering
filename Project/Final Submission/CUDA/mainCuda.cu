#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <string>
using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::string;
using std::setprecision;



__device__ double selectKth(double* arr,int * index, int k, int len, int startIndex) {
// double selectKth(double* arr, int k, int len) {
 
 int from = startIndex, to = (startIndex+len) - 1;
 
 // if from == to we reached the kth element
 while (from < to) {
  int r = from, w = to;
  double mid = arr[(r + w) / 2];
 
  // stop if the reader and writer meets
  while (r < w) {
 
   if (arr[r] >= mid) { // put the large values at the end
    double tmp = arr[w];
    int tempIndex=index[w];
    arr[w] = arr[r];
    index[w]=index[r];
    index[r]=tempIndex;
    arr[r] = tmp;
    w--;
   } else { // the value is smaller than the pivot, skip
    r++;
   }
  }
 
  // if we stepped up (r++) we need to step one down
  if (arr[r] > mid)
   r--;
 
  // the r pointer is on the end of the first k elements
  if ((k+startIndex) <= r) {
   to = r;
  } else {
   from = r + 1;
  }
 }
 
 return arr[k+startIndex];
}


//double* sort(double * array,int* indexArray, int size, int N ){
__device__ void sort(double * array,int* indexArray, int size, int N, int startIndex ){

 // double * dupplicate = malloc(sizeof(double)*size);
 //  for(int i=0;i<size;i++){
 //    dupplicate[i]=array[i];
 //  }
  //double *  unsorted = malloc(sizeof(double)*N);
  //int * unsoretedIndex =malloc(sizeof(int)*N);
  // double kth = selectKth(dupplicate,N-1,size);
  double kth = selectKth(array, indexArray, N-1, size, startIndex);
  //printf("Start Index: %d\t Kth Value: %f\n",startIndex,kth );
  
  //printf("\n");
  //printf("\n");
  //printf("Kth Element : %f\n",kth );
  int j=startIndex;
  int i=startIndex;
  for(i=startIndex; i<startIndex+size; i++){
    if(array[i]<kth){

      array[j] = array[i];
      indexArray[j] = indexArray[i];
      //unsoretedIndex[j]=indexArray[i];
      //printf("unsorted in loop : %f\n",unsorted[j] );
      j++;
    }
  }
  //printf("outside for J:%d\n",j);
  while(j<N){
     //printf("J: %d\n",j);
     array[j]=kth;
     //printf("Inside while: Unsorted is : %f\n",unsorted[j]);
     j++;
  }
  //return unsorted;

}

__global__ void distance(const int nodeNumber,const int * x, const int * y,double * distance, int * index) {
		//CODE MOTION

		unsigned int first = (blockIdx.x * blockDim.x) + threadIdx.x;
		unsigned int second = (blockIdx.y * blockDim.y) + threadIdx.y;
		if(first<nodeNumber && second<nodeNumber){
			int x1 = x[first];
			int y1 = y[first];
			double sq1 = (x1-x[second])*(x1-x[second]);
			double sq2 = (y1-y[second])*(y1-y[second]);
			distance[(first*nodeNumber)+second]=sq1 + sq2;
			index[(first*nodeNumber)+second]=second;
			//printf("Distance Calculated for i : %d\t%f\n", );
			//nodes[first].neighbourID[second]=second;
		}

	
}
__global__ void test2( int N, const int nodeNumber, double * distance, int * indexArray) {
		unsigned int first = blockIdx.x * blockDim.x + threadIdx.x;//* blockDim.y+ threadIdx.y * blockDim.x 
		//unsigned int second = (blockIdx.y * blockDim.y) + threadIdx.y;
		
		
		if(first<nodeNumber){
			//printf("FIRST: %d\n",first);
			sort(distance, indexArray, nodeNumber, N, (first*nodeNumber));
			//printf("Distance Calculated for i : %d\t%f\n", );
			//nodes[first].neighbourID[second]=second;
		}
}




// int main(int argc, char **argv)
// {
  
//   int index[10]={0,1,2,3,4,5,6,7,8,9};
//   double arr[10] ={0,3,10,9,7,5,1,6,2,0};
//   //printf("\n");

//   double* kth = sort(arr,index,10,5);
//   //printf("returned \n");
//   for(int i=0;i<5;i++){
//     //printf("Index: %d \t Value: %f\n",index[i],kth[i]);
//   }
//   //printf("\n");
  
//   return 0;
// }
void distanceMain(const int N,const int nodeNumber,const int * x, const int * y,double * distanceVar, int * index) {
	   //Memory Timer		    
	NSTimer memoryTimer("MemoryTimer", false, false);
		//Creating variables to be used by Kernel
	int  * d_x;
	int * d_y;
	int * d_index;
	double * d_distance;

	//unsigned char* d_out;
	//Allocating memory on Device
	hipMalloc((void**) &d_x, (nodeNumber*sizeof(int)));
	hipMalloc((void**) &d_y, (nodeNumber*sizeof(int)));
	hipMalloc((void**) &d_distance, (nodeNumber*nodeNumber*sizeof(double)));
	hipMalloc((void**) &d_index, (nodeNumber*nodeNumber*sizeof(int)));
	//hipMalloc((void**) &d_out,width*height);
	memoryTimer.start();
	//Copying from Host to device 
	hipMemcpy(d_x, x, (nodeNumber*sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, (nodeNumber*sizeof(int)), hipMemcpyHostToDevice);
	memoryTimer.stop();
	 //Number of blocks is (width/16)+1 and (height/16)+1. We add 1 to each in case the width or height is not perfectly divisible by 1. We cater for any extra 
	 // threads we get in the kernel
	dim3 gridSize((nodeNumber/16)+1,(nodeNumber/16)+1);
		//16x16 Threads per block (utalizing 256 threads per block)
	dim3 blockSize(16,16);
	//Kernel Timer
	NSTimer kernelTimer = NSTimer("darker", false, false);
	kernelTimer.start();
		// Starting Kernel
	distance<<< gridSize, blockSize >>>(nodeNumber,d_x,d_y,d_distance,d_index);
	hipDeviceSynchronize();

	dim3 gridSize2((nodeNumber/256)+1,(nodeNumber/1)+1);
    dim3 blockSize2(1,1);
	test2<<<256, 256 >>>(N,nodeNumber,d_distance,d_index);
	hipDeviceSynchronize();
	kernelTimer.stop();
			  
	memoryTimer.start();
	// Copying Resylts back from Device To Host
	hipMemcpy(distanceVar, d_distance, (nodeNumber*nodeNumber*sizeof(double)), hipMemcpyDeviceToHost);
	hipMemcpy(index, d_index, (nodeNumber*nodeNumber*sizeof(int)), hipMemcpyDeviceToHost);
	memoryTimer.stop();
	//Printing Time
	cout << "Kernel (s): \t"<<fixed <<setprecision(6)<< kernelTimer.getElapsed() << endl;
  	cout << "Memory (s): \t"<<fixed <<setprecision(6)<< memoryTimer.getElapsed() << endl;


}